#include "hip/hip_runtime.h"
%%writefile phonebook_search.cu
#include <bits/stdc++.h>
using namespace std;
#include <hip/hip_runtime.h>

struct Contact{
    char name[65];
    char phone_number[65];
};


string getInput(ifstream& file){
    string ans;
    char c;
    int readSuru = 0;
    while(file.get(c)){
        if(c == '\"'){
            if(readSuru == 1) break;
            readSuru = 1;
        }else{
            if(readSuru){
                ans.push_back(c);
            }
        }
    }
    return ans;
}

__device__ bool check(char* str1, char* str2){
    for(int i = 0; str1[i] != '\0'; i++){
        int flag = 1;
        for(int j = 0; str2[j] != '\0' ; j++){
            if(str1[i + j] != str2[j]){
                flag = 0;
                break;
            }
        }
        if(flag == 1) return true;
    }
    return false;
}


__global__ void myKernel(Contact* phoneBook, char* pat, int offset){
    int threadNumber = threadIdx.x + offset;
    if(check(phoneBook[threadNumber].name, pat)){
        printf("%s %s\n", phoneBook[threadNumber].name, phoneBook[threadNumber].phone_number);
    }
}



int main(int argc, char* argv[])
{
    int threadLimit = atoi(argv[2]);

    ifstream myfile("one.txt");
    vector<Contact> phoneBook;

    int count = 0;

    while(myfile.peek() != EOF){

        if(count > 10000) break;
        count++;

        string name = getInput(myfile);
        string phoneNum = getInput(myfile);

        Contact c;
        strcpy(c.name, name.c_str());
        strcpy(c.phone_number, phoneNum.c_str());

        phoneBook.push_back(c);
    }

    string search_name = argv[1];
    char pat[65];
    strcpy(pat, search_name.c_str());


    char* d_pat;
    hipMalloc(&d_pat, 65); //memory allocation
    hipMemcpy(d_pat, pat, 65, hipMemcpyHostToDevice); //copying to device

    int n = phoneBook.size();
    Contact* d_phoneBook;
    hipMalloc(&d_phoneBook, n*sizeof(Contact));
    hipMemcpy(d_phoneBook, phoneBook.data(), n * sizeof(Contact), hipMemcpyHostToDevice);


    int bakiAche = n;
    int offset = 0;
    while(bakiAche > 0){
        int batchSize = min(threadLimit, bakiAche);
        myKernel<<<1,batchSize>>>(d_phoneBook, d_pat, offset);
        hipDeviceSynchronize();

        bakiAche -= batchSize;
        offset += batchSize;
    }

}
